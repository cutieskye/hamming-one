#include "hip/hip_runtime.h"
﻿#include "common.h"

#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <iostream>

__global__ void findHammingOne(int* data, const int N, const int L)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = tid + 1; i < N; ++i) {
		int distance = 0;
		for (int j = 0; j < L && distance <= 1; ++j) {
			int diff = data[tid * L + j] ^ data[i * L + j];
			if (diff) {
				if (diff & (diff - 1)) {
					distance = INT_MAX;
				}
				else {
					++distance;
				}
			}
		}
		if (distance == 1) {
			printf("(%d, %d)\n", tid, i);
		}
	}
}

void readFile(int*& data, int& N, int& L, const std::string fileName)
{
	constexpr int bits = CHAR_BIT * sizeof(*data);
	std::ifstream file(fileName);
	if (!file.is_open()) {
		fprintf(stderr, "ifstream failed!\n");
		exit(EXIT_FAILURE);
	}
	file >> N >> L;
	int newL = (int)ceil((double)L / bits);
	cudaCheckErrors(hipMallocManaged((void**)&data, N * L * sizeof(*data)));
	memset(data, 0, N * newL * sizeof(*data));
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < newL; ++j) {
			for (int k = 0; k < bits && j * bits + k < L; ++k) {
				char b;
				file >> b;
				if (b == '0') {
					data[i * newL + j] = data[i * newL + j] << 1;
				}
				else if (b == '1') {
					data[i * newL + j] = (data[i * newL + j] << 1) + 1;
				}
			}
		}
	}
	L = newL;
}

int main()
{
	constexpr int device = 0;

	cudaCheckErrors(hipSetDevice(device));
	cudaCheckErrors(hipDeviceSetLimit(hipLimitPrintfFifoSize, ULONG_MAX));

	hipEvent_t start{};
	hipEvent_t end{};

	cudaCheckErrors(hipEventCreate(&start));
	cudaCheckErrors(hipEventCreate(&end));

	const std::string fileName = "hamming_one.txt";
	int N = 0;
	int L = 0;
	int* data = nullptr;

	readFile(data, N, L, fileName);

	hipDeviceProp_t prop{};

	cudaCheckErrors(hipGetDeviceProperties(&prop, device));

	int numThreads = prop.maxThreadsPerBlock;
	int numBlocks = (int)ceil((double)N / numThreads);

	cudaCheckErrors(hipEventRecord(start));
	findHammingOne << <numBlocks, numThreads >> > (data, N, L);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(end));
	cudaCheckErrors(hipEventSynchronize(end));

	float time = .0f;

	cudaCheckErrors(hipEventElapsedTime(&time, start, end));

	printf("Searching for pairs with the Hamming distance equal to one took %f ms\n", time);

	cudaCheckErrors(hipEventDestroy(start));
	cudaCheckErrors(hipEventDestroy(end));
	cudaCheckErrors(hipFree(data));
	cudaCheckErrors(hipDeviceReset());
	return EXIT_SUCCESS;
}