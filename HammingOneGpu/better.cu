#include "hip/hip_runtime.h"
﻿#include "common.h"

#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <fstream>

#include <thrust/execution_policy.h>
#include <thrust/pair.h>
#include <thrust/sort.h>

// Build a polynomial rolling hash
__global__ void computeHashes(thrust::pair<uint_fast64_t, int>* const hashes, const char* d_data, const int N, const int L, const uint_fast64_t multiplier, const uint_fast64_t modulus)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		uint_fast64_t hash = 0;
		uint_fast64_t a = 1;
		for (int i = 0; i < L; ++i) {
			hash = (hash + a * d_data[tid * L + i]) % modulus;
			a = a * multiplier % modulus;
		}
		thrust::pair<uint_fast64_t, int> el{ hash, tid };
		// Allow multiple entries with the same key (multimap)
		hashes[tid] = el;
	}
}

// Return the index of the rightmost element if there are duplicates
__device__ int binarySearch(const thrust::pair<uint_fast64_t, int>* const hashes, const uint_fast64_t newHash, const int N)
{
	int left = 0;
	int right = N - 1;
	while (left != right) {
		int middle = (int)ceil(((double)left + right) / 2);
		if (hashes[middle].first > newHash) {
			right = middle - 1;
		}
		else {
			left = middle;
		}
	}
	if (hashes[left].first == newHash) {
		return left;
	}
	return -1;
}

// Find pairs with the Hamming distance equal to one in O(N*L*log(N)) time
__global__ void findHammingOne(const thrust::pair<uint_fast64_t, int>* const hashes, const char* const d_data, const int N, const int L, const uint_fast64_t multiplier, const uint_fast64_t modulus)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		uint_fast64_t a = 1;
		for (int i = 0; i < L; ++i) {
			thrust::pair<uint_fast64_t, int> el = hashes[tid];
			// Update the hash in constant time
			int s = d_data[el.second * L + i] == 1 ? 1 : -1;
			uint_fast64_t newHash = (el.first + s * a + modulus) % modulus;
			int j = binarySearch(hashes, newHash, N);
			while (j >= 0) {
				thrust::pair<uint_fast64_t, int> newEl = hashes[j];
				if (newEl.first != newHash) {
					break;
				}
				// Print pairs only once
				if (el.second < newEl.second) {
					printf("(%d, %d)\n", el.second, newEl.second);
				}
				--j;
			}
			a = a * multiplier % modulus;
		}
	}
}

// Read data from an existing well-formatted file
void readFile(char*& h_data, int& N, int& L, const std::string fileName)
{
	std::ifstream file(fileName);
	if (!file.is_open()) {
		fprintf(stderr, "ifstream failed!\n");
		exit(EXIT_FAILURE);
	}
	file >> N >> L;
	h_data = new char[N * L];
	if (!h_data) {
		fprintf(stderr, "new failed!\n");
		exit(EXIT_FAILURE);
	}
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < L; ++j) {
			char b;
			file >> b;
			if (b == '0') {
				// Replace 0's with 2's to get good hashing
				h_data[i * L + j] = 2;
			}
			else if (b == '1') {
				h_data[i * L + j] = 1;
			}
		}
	}
}

// Write time measurements to a unique file
void writeStats(const int N, const int L, const float readTime, const float memcpyTime, const float computeTime, const float sortTime, const float findTime)
{
	std::string now = std::to_string(std::chrono::system_clock::to_time_t(std::chrono::system_clock::now()));
	std::ofstream file("stats_" + now + ".txt");
	if (!file.is_open()) {
		fprintf(stderr, "ofstream failed!\n");
		exit(1);
	}
	file << "Reading " << N << " binary sequences of length " << L << ": " << readTime
		<< " ms\nCopying data from host to device memory: " << memcpyTime
		<< " ms\nComputing hashes: " << computeTime
		<< " ms\nSorting hashes: " << sortTime
		<< " ms\nSearching for pairs with the Hamming distance equal to one: " << findTime << " ms\n";
}

int main()
{
	constexpr int device = 0;

	cudaCheckErrors(hipSetDevice(device));
	cudaCheckErrors(hipDeviceSetLimit(hipLimitPrintfFifoSize, ULONG_MAX));

	hipEvent_t readStart{};
	hipEvent_t readEnd{};
	hipEvent_t memcpyStart{};
	hipEvent_t memcpyEnd{};
	hipEvent_t computeStart{};
	hipEvent_t computeEnd{};
	hipEvent_t sortStart{};
	hipEvent_t sortEnd{};
	hipEvent_t findStart{};
	hipEvent_t findEnd{};

	cudaCheckErrors(hipEventCreate(&readStart));
	cudaCheckErrors(hipEventCreate(&readEnd));
	cudaCheckErrors(hipEventCreate(&memcpyStart));
	cudaCheckErrors(hipEventCreate(&memcpyEnd));
	cudaCheckErrors(hipEventCreate(&computeStart));
	cudaCheckErrors(hipEventCreate(&computeEnd));
	cudaCheckErrors(hipEventCreate(&sortStart));
	cudaCheckErrors(hipEventCreate(&sortEnd));
	cudaCheckErrors(hipEventCreate(&findStart));
	cudaCheckErrors(hipEventCreate(&findEnd));

	const std::string fileName = "hamming_one.txt";
	int N = 0;
	int L = 0;
	char* h_data = nullptr;

	cudaCheckErrors(hipEventRecord(readStart));
	readFile(h_data, N, L, fileName);
	cudaCheckErrors(hipEventRecord(readEnd));

	char* d_data = nullptr;

	cudaCheckErrors(hipMalloc((void**)&d_data, N * L * sizeof(*d_data)));

	cudaCheckErrors(hipEventRecord(memcpyStart));
	cudaCheckErrors(hipMemcpy(d_data, h_data, N * L * sizeof(*d_data), hipMemcpyHostToDevice));
	cudaCheckErrors(hipEventRecord(memcpyEnd));

	thrust::pair<uint_fast64_t, int>* hashes = nullptr;

	cudaCheckErrors(hipMalloc((void**)&hashes, N * sizeof(*hashes)));

	hipDeviceProp_t prop{};

	cudaCheckErrors(hipGetDeviceProperties(&prop, device));

	int numThreads = prop.maxThreadsPerBlock;
	int numBlocks = (int)ceil((double)N / numThreads);
	constexpr uint_fast64_t multiplier = 16807;
	constexpr uint_fast64_t modulus = 2147483647;

	cudaCheckErrors(hipEventRecord(computeStart));
	computeHashes << <numBlocks, numThreads >> > (hashes, d_data, N, L, multiplier, modulus);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(computeEnd));

	cudaCheckErrors(hipEventRecord(sortStart));
	// Sort in ascending order (the first element of a pair is compared first)
	thrust::sort(thrust::device, hashes, hashes + N);
	cudaCheckErrors(hipEventRecord(sortEnd));

	cudaCheckErrors(hipEventRecord(findStart));
	findHammingOne << <numBlocks, numThreads >> > (hashes, d_data, N, L, multiplier, modulus);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(findEnd));
	cudaCheckErrors(hipEventSynchronize(findEnd));

	float readTime = .0f;
	float memcpyTime = .0f;
	float computeTime = .0f;
	float sortTime = .0f;
	float findTime = .0f;

	cudaCheckErrors(hipEventElapsedTime(&readTime, readStart, readEnd));
	cudaCheckErrors(hipEventElapsedTime(&memcpyTime, memcpyStart, memcpyEnd));
	cudaCheckErrors(hipEventElapsedTime(&computeTime, computeStart, computeEnd));
	cudaCheckErrors(hipEventElapsedTime(&sortTime, sortStart, sortEnd));
	cudaCheckErrors(hipEventElapsedTime(&findTime, findStart, findEnd));

	writeStats(N, L, readTime, memcpyTime, computeTime, sortTime, findTime);

	delete[] h_data;
	cudaCheckErrors(hipEventDestroy(readStart));
	cudaCheckErrors(hipEventDestroy(readEnd));
	cudaCheckErrors(hipEventDestroy(memcpyStart));
	cudaCheckErrors(hipEventDestroy(memcpyEnd));
	cudaCheckErrors(hipEventDestroy(computeStart));
	cudaCheckErrors(hipEventDestroy(computeEnd));
	cudaCheckErrors(hipEventDestroy(sortStart));
	cudaCheckErrors(hipEventDestroy(sortEnd));
	cudaCheckErrors(hipEventDestroy(findStart));
	cudaCheckErrors(hipEventDestroy(findEnd));
	cudaCheckErrors(hipFree(d_data));
	cudaCheckErrors(hipFree(hashes));
	cudaCheckErrors(hipDeviceReset());
	return EXIT_SUCCESS;
}