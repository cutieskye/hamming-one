#include "common.h"

#include <fstream>

void check(const hipError_t cudaStatus, const char* func, const char* file, const int line) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", file, line, cudaStatus, hipGetErrorString(cudaStatus), func);
		exit(EXIT_FAILURE);
	}
}