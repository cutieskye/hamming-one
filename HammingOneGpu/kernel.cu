#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <fstream>

#include <thrust/execution_policy.h>
#include <thrust/pair.h>
#include <thrust/sort.h>

// check for runtime errors when calling CUDA functions
void check(const hipError_t cudaStatus, const char* func, const char* file, const int line) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", file, line, cudaStatus, hipGetErrorString(cudaStatus), func);
		exit(EXIT_FAILURE);
	}
}

#define cudaCheckErrors(val) check((val), #val, __FILE__, __LINE__);

// build a polynomial rolling hash
__global__ void computeHashes(thrust::pair<uint_fast64_t, int>* const hashes, const char* d_data, const int N, const int L, const uint_fast64_t multiplier, const uint_fast64_t modulus)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		uint_fast64_t hash = 0;
		uint_fast64_t a = 1;
		for (int i = 0; i < L; ++i) {
			hash = (hash + a * d_data[tid * L + i]) % modulus;
			a = a * multiplier % modulus;
		}
		thrust::pair<uint_fast64_t, int> el{ hash, tid };
		// allow multiple entries with the same key (multimap)
		hashes[tid] = el;
	}
}

// return the index of the rightmost element if there are duplicates
__device__ int binarySearch(const thrust::pair<uint_fast64_t, int>* const hashes, const uint_fast64_t newHash, const int N)
{
	int left = 0;
	int right = N - 1;
	while (left != right) {
		int middle = (int)ceil(((double)left + right) / 2);
		if (hashes[middle].first > newHash) {
			right = middle - 1;
		}
		else {
			left = middle;
		}
	}
	if (hashes[left].first == newHash) {
		return left;
	}
	return -1;
}

// find pairs with the Hamming distance equal to one in O(N*L*log(N)) time
__global__ void findHammingOne(const thrust::pair<uint_fast64_t, int>* const hashes, const char* const d_data, const int N, const int L, const uint_fast64_t multiplier, const uint_fast64_t modulus)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		uint_fast64_t a = 1;
		for (int i = 0; i < L; ++i) {
			thrust::pair<uint_fast64_t, int> el = hashes[tid];
			// update the hash in constant time
			int s = d_data[el.second * L + i] == 1 ? 1 : -1;
			uint_fast64_t newHash = (el.first + s * a + modulus) % modulus;
			int j = binarySearch(hashes, newHash, N);
			while (j >= 0) {
				thrust::pair<uint_fast64_t, int> newEl = hashes[j];
				if (newEl.first != newHash) {
					break;
				}
				// print pairs only once
				if (el.second < newEl.second) {
					printf("(%d, %d)\n", el.second, newEl.second);
				}
				--j;
			}
			a = a * multiplier % modulus;
		}
	}
}

// read data from an existing well-formatted file
void readFile(char*& h_data, int& N, int& L, const std::string fileName)
{
	std::ifstream file(fileName);
	if (!file.is_open()) {
		fprintf(stderr, "ifstream failed!\n");
		exit(EXIT_FAILURE);
	}
	file >> N >> L;
	h_data = new char[N * L];
	if (!h_data) {
		fprintf(stderr, "new failed!\n");
		exit(EXIT_FAILURE);
	}
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < L; ++j) {
			char b;
			file >> b;
			if (b == '0') {
				// replace 0's with 2's to get good hashing
				h_data[i * L + j] = 2;
			}
			else if (b == '1') {
				h_data[i * L + j] = 1;
			}
		}
	}
}

// write time measurements to a unique file
void writeStats(const int N, const int L, const float readTime, const float memcpyTime, const float computeTime, const float sortTime, const float findTime)
{
	std::string now = std::to_string(std::chrono::system_clock::to_time_t(std::chrono::system_clock::now()));
	std::ofstream file("stats_" + now + ".txt");
	if (!file.is_open()) {
		fprintf(stderr, "ofstream failed!\n");
		exit(1);
	}
	file << "Reading " << N << " binary sequences of length " << L << ": " << readTime
		<< " ms\nCopying data from host to device memory: " << memcpyTime
		<< " ms\nComputing hashes: " << computeTime
		<< " ms\nSorting hashes: " << sortTime
		<< " ms\nSearching for pairs with the Hamming distance equal to one: " << findTime << " ms";
}

int main()
{
	cudaCheckErrors(hipSetDevice(0));
	cudaCheckErrors(hipDeviceSetLimit(hipLimitPrintfFifoSize, ULONG_MAX));

	hipEvent_t readStart{};
	hipEvent_t readEnd{};
	hipEvent_t memcpyStart{};
	hipEvent_t memcpyEnd{};
	hipEvent_t computeStart{};
	hipEvent_t computeEnd{};
	hipEvent_t sortStart{};
	hipEvent_t sortEnd{};
	hipEvent_t findStart{};
	hipEvent_t findEnd{};

	cudaCheckErrors(hipEventCreate(&readStart));
	cudaCheckErrors(hipEventCreate(&readEnd));
	cudaCheckErrors(hipEventCreate(&memcpyStart));
	cudaCheckErrors(hipEventCreate(&memcpyEnd));
	cudaCheckErrors(hipEventCreate(&computeStart));
	cudaCheckErrors(hipEventCreate(&computeEnd));
	cudaCheckErrors(hipEventCreate(&sortStart));
	cudaCheckErrors(hipEventCreate(&sortEnd));
	cudaCheckErrors(hipEventCreate(&findStart));
	cudaCheckErrors(hipEventCreate(&findEnd));

	const std::string fileName = "hamming_one.txt";
	int N = 0;
	int L = 0;
	char* h_data = nullptr;

	cudaCheckErrors(hipEventRecord(readStart));
	readFile(h_data, N, L, fileName);
	cudaCheckErrors(hipEventRecord(readEnd));

	char* d_data = nullptr;

	cudaCheckErrors(hipMalloc((void**)&d_data, N * L * sizeof(*d_data)));

	cudaCheckErrors(hipEventRecord(memcpyStart));
	cudaCheckErrors(hipMemcpy(d_data, h_data, N * L * sizeof(*d_data), hipMemcpyHostToDevice));
	cudaCheckErrors(hipEventRecord(memcpyEnd));

	thrust::pair<uint_fast64_t, int>* hashes = nullptr;

	cudaCheckErrors(hipMalloc((void**)&hashes, N * sizeof(*hashes)));

	hipDeviceProp_t prop{};

	cudaCheckErrors(hipGetDeviceProperties(&prop, 0));

	int numThreads = prop.maxThreadsPerBlock;
	int numBlocks = (int)ceil((double)N / numThreads);
	constexpr uint_fast64_t multiplier = 16807;
	constexpr uint_fast64_t modulus = 2147483647;

	cudaCheckErrors(hipEventRecord(computeStart));
	computeHashes << <numBlocks, numThreads >> > (hashes, d_data, N, L, multiplier, modulus);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(computeEnd));

	cudaCheckErrors(hipEventRecord(sortStart));
	thrust::sort(thrust::device, hashes, hashes + N);
	cudaCheckErrors(hipEventRecord(sortEnd));

	cudaCheckErrors(hipEventRecord(findStart));
	findHammingOne << <numBlocks, numThreads >> > (hashes, d_data, N, L, multiplier, modulus);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(findEnd));
	cudaCheckErrors(hipEventSynchronize(findEnd));

	float readTime = .0f;
	float memcpyTime = .0f;
	float computeTime = .0f;
	float sortTime = .0f;
	float findTime = .0f;

	cudaCheckErrors(hipEventElapsedTime(&readTime, readStart, readEnd));
	cudaCheckErrors(hipEventElapsedTime(&memcpyTime, memcpyStart, memcpyEnd));
	cudaCheckErrors(hipEventElapsedTime(&computeTime, computeStart, computeEnd));
	cudaCheckErrors(hipEventElapsedTime(&sortTime, sortStart, sortEnd));
	cudaCheckErrors(hipEventElapsedTime(&findTime, findStart, findEnd));

	writeStats(N, L, readTime, memcpyTime, computeTime, sortTime, findTime);

	delete[] h_data;
	cudaCheckErrors(hipEventDestroy(readStart));
	cudaCheckErrors(hipEventDestroy(readEnd));
	cudaCheckErrors(hipEventDestroy(memcpyStart));
	cudaCheckErrors(hipEventDestroy(memcpyEnd));
	cudaCheckErrors(hipEventDestroy(computeStart));
	cudaCheckErrors(hipEventDestroy(computeEnd));
	cudaCheckErrors(hipEventDestroy(sortStart));
	cudaCheckErrors(hipEventDestroy(sortEnd));
	cudaCheckErrors(hipEventDestroy(findStart));
	cudaCheckErrors(hipEventDestroy(findEnd));
	cudaCheckErrors(hipFree(d_data));
	cudaCheckErrors(hipFree(hashes));
	cudaCheckErrors(hipDeviceReset());
	return EXIT_SUCCESS;
}