#include "hip/hip_runtime.h"
﻿#include "common.h"

#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <iostream>

#include <hiprand/hiprand_kernel.h>

__global__ void generateData(bool* data, const int N, const int L, const int K)
{
	const int size = N * L;
	const int part = size * K / 10;
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state{};
	hiprand_init((unsigned long long)clock(), tid, 0, &state);
	if (tid < size - part) {
		const bool a = hiprand(&state) % 2;
		data[tid] = a;
		if (tid < part) {
			data[size - part + tid] = a;
		}
	}
}

__global__ void mutateData(bool* data, const int N, const int L, const int K)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state{};
	hiprand_init((unsigned long long)clock(), tid, 0, &state);
	if (tid < N * K / 10) {
		int pos = hiprand(&state) % L;
		data[L * tid + pos] ^= 1;
	}
}

void writeData(const std::string fileName, const bool* a, const int N, const int L) {
	std::ofstream file(fileName);
	if (!file.is_open()) {
		fprintf(stderr, "ofstream failed!");
		exit(EXIT_FAILURE);
	}
	file << N << " " << L << "\n";
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < L; ++j) {
			file << a[i * L + j];
		}
		file << "\n";
	}
}

int main()
{
	constexpr int device = 0;

	// Choose which GPU to run on, change this on a multi-GPU system
	cudaCheckErrors(hipSetDevice(device));

	constexpr int N = 100'000;
	constexpr int L = 1'000;

	bool* data = nullptr;

	// Allocate a GPU buffer for one vector
	cudaCheckErrors(hipMallocManaged((void**)&data, N * L * sizeof(*data)));

	hipEvent_t start{};
	hipEvent_t end{};

	cudaCheckErrors(hipEventCreate(&start));
	cudaCheckErrors(hipEventCreate(&end));

	hipDeviceProp_t prop{};

	cudaCheckErrors(hipGetDeviceProperties(&prop, device));

	int numThreads = prop.maxThreadsPerBlock;
	int numBlocks = (int)ceil((double)N * L / numThreads);
	constexpr int K = 3;

	cudaCheckErrors(hipEventRecord(start));

	// Launch a kernel on the GPU
	generateData << <numBlocks, numThreads >> > (data, N, L, K);

	// Check for any errors launching the kernel
	cudaCheckErrors(hipGetLastError());

	// Launch a kernel on the GPU
	numBlocks = (int)ceil(N * K / 10.0 / numThreads);
	mutateData << <numBlocks, numThreads >> > (data, N, L, K);

	// Check for any errors launching the kernel
	cudaCheckErrors(hipGetLastError());

	cudaCheckErrors(hipEventRecord(end));
	cudaCheckErrors(hipEventSynchronize(end));

	const std::string fileName = "hamming_one.txt";

	writeData(fileName, data, N, L);

	float time = .0f;

	cudaCheckErrors(hipEventElapsedTime(&time, start, end));

	std::cout << "Generating " << N << " binary sequences of length " << L << " took " << time << " ms\n";

	cudaCheckErrors(hipEventDestroy(start));
	cudaCheckErrors(hipEventDestroy(end));

	cudaCheckErrors(hipFree(data));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces
	cudaCheckErrors(hipDeviceReset());

	return EXIT_SUCCESS;
}