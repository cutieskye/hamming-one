#include "hip/hip_runtime.h"
﻿#include "common.h"

#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <fstream>

#include <thrust/execution_policy.h>
#include <thrust/pair.h>
#include <thrust/sort.h>

constexpr uint_fast64_t firstMultiplier = 16807;
constexpr uint_fast64_t secondMultiplier = 8121;
constexpr uint_fast64_t firstModulus = 2147483647;
constexpr uint_fast64_t secondModulus = 2305843009213693951;

// Build a polynomial rolling hash
__global__ void computeHashes(thrust::pair<thrust::pair<uint_fast64_t, uint_fast64_t>, int>* const hashes, const char* d_data, const int N, const int L)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		uint_fast64_t firstHash = 0;
		uint_fast64_t secondHash = 0;
		uint_fast64_t firstCoefficient = firstMultiplier;
		uint_fast64_t secondCoefficient = 1;
		for (int i = 0; i < L; ++i) {
			firstHash = (firstHash + firstCoefficient * d_data[tid * L + i]) % firstModulus;
			secondHash = (secondHash + secondCoefficient * d_data[tid * L + i]) % secondModulus;
			firstCoefficient = firstCoefficient * firstMultiplier % firstModulus;
			secondCoefficient = secondCoefficient * secondMultiplier % secondModulus;
		}
		thrust::pair< thrust::pair<uint_fast64_t, uint_fast64_t>, int> el{ {firstHash, secondHash}, tid };
		// Allow multiple entries with the same key (multimap)
		hashes[tid] = el;
	}
}

// Return the index of the rightmost element if there are duplicates
__device__ int binarySearch(const thrust::pair<thrust::pair<uint_fast64_t, uint_fast64_t>, int>* const hashes, const thrust::pair<uint_fast64_t, uint_fast64_t> newHashPair, const int N)
{
	int left = 0;
	int right = N - 1;
	thrust::pair<uint_fast64_t, uint_fast64_t> hashPair{};
	while (left != right) {
		int middle = (int)ceil(((double)left + right) / 2);
		hashPair = hashes[middle].first;
		if (hashPair.first > newHashPair.first
			|| (hashPair.first == newHashPair.first && hashPair.second > newHashPair.second)) {
			right = middle - 1;
		}
		else {
			left = middle;
		}
	}
	hashPair = hashes[left].first;
	if (hashPair.first == newHashPair.first || hashPair.second == newHashPair.second) {
		return left;
	}
	return -1;
}

// Find pairs with the Hamming distance equal to one in O(N*L*log(N)) time
__global__ void findHammingOne(const thrust::pair<thrust::pair<uint_fast64_t, uint_fast64_t>, int>* const hashes, const char* const d_data, const int N, const int L)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N) {
		const thrust::pair<thrust::pair<uint_fast64_t, uint_fast64_t>, int> el = hashes[tid];
		uint_fast64_t firstCoefficient = firstMultiplier;
		uint_fast64_t secondCoefficient = 1;
		for (int i = 0; i < L; ++i) {
			// Update the hash in constant time
			int s = d_data[el.second * L + i] == 1 ? 1 : -1;
			thrust::pair<uint_fast64_t, uint_fast64_t> newHashPair{
				(el.first.first + s * firstCoefficient + firstModulus) % firstModulus,
				(el.first.second + s * secondCoefficient + secondModulus) % secondModulus
			};
			int j = binarySearch(hashes, newHashPair, N);
			while (j >= 0) {
				thrust::pair<thrust::pair<uint_fast64_t, uint_fast64_t>, int> newEl = hashes[j];
				if (newEl.first.first != newHashPair.first || newEl.first.second != newHashPair.second) {
					break;
				}
				// Print pairs only once
				if (el.second < newEl.second) {
					printf("(%d, %d)\n", el.second, newEl.second);
				}
				--j;
			}
			firstCoefficient = firstCoefficient * firstMultiplier % firstModulus;
			secondCoefficient = secondCoefficient * secondMultiplier % secondModulus;
		}
	}
}

// Read data from an existing well-formatted file
void readFile(char*& h_data, int& N, int& L, const std::string fileName)
{
	std::ifstream file(fileName);
	if (!file.is_open()) {
		fprintf(stderr, "ifstream failed!\n");
		exit(EXIT_FAILURE);
	}
	file >> N >> L;
	h_data = new char[N * L];
	if (!h_data) {
		fprintf(stderr, "new failed!\n");
		exit(EXIT_FAILURE);
	}
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < L; ++j) {
			char b;
			file >> b;
			if (b == '0') {
				// Replace 0's with 2's to get good hashing
				h_data[i * L + j] = 2;
			}
			else if (b == '1') {
				h_data[i * L + j] = 1;
			}
		}
	}
}

// Write time measurements to a unique file
void writeStats(const int N, const int L, const float readTime, const float memcpyTime, const float computeTime, const float sortTime, const float findTime)
{
	std::string now = std::to_string(std::chrono::system_clock::to_time_t(std::chrono::system_clock::now()));
	std::ofstream file("stats_" + now + ".txt");
	if (!file.is_open()) {
		fprintf(stderr, "ofstream failed!\n");
		exit(1);
	}
	file << "Reading " << N << " binary sequences of length " << L << ": " << readTime
		<< " ms\nCopying data from host to device memory: " << memcpyTime
		<< " ms\nComputing hashes: " << computeTime
		<< " ms\nSorting hashes: " << sortTime
		<< " ms\nSearching for pairs with the Hamming distance equal to one: " << findTime << " ms\n";
}

int main()
{
	constexpr int device = 0;

	cudaCheckErrors(hipSetDevice(device));
	cudaCheckErrors(hipDeviceSetLimit(hipLimitPrintfFifoSize, ULONG_MAX));

	hipEvent_t readStart{};
	hipEvent_t readEnd{};
	hipEvent_t memcpyStart{};
	hipEvent_t memcpyEnd{};
	hipEvent_t computeStart{};
	hipEvent_t computeEnd{};
	hipEvent_t sortStart{};
	hipEvent_t sortEnd{};
	hipEvent_t findStart{};
	hipEvent_t findEnd{};

	cudaCheckErrors(hipEventCreate(&readStart));
	cudaCheckErrors(hipEventCreate(&readEnd));
	cudaCheckErrors(hipEventCreate(&memcpyStart));
	cudaCheckErrors(hipEventCreate(&memcpyEnd));
	cudaCheckErrors(hipEventCreate(&computeStart));
	cudaCheckErrors(hipEventCreate(&computeEnd));
	cudaCheckErrors(hipEventCreate(&sortStart));
	cudaCheckErrors(hipEventCreate(&sortEnd));
	cudaCheckErrors(hipEventCreate(&findStart));
	cudaCheckErrors(hipEventCreate(&findEnd));

	const std::string fileName = "hamming_one.txt";
	int N = 0;
	int L = 0;
	char* h_data = nullptr;

	cudaCheckErrors(hipEventRecord(readStart));
	readFile(h_data, N, L, fileName);
	cudaCheckErrors(hipEventRecord(readEnd));

	char* d_data = nullptr;

	cudaCheckErrors(hipMalloc((void**)&d_data, N * L * sizeof(*d_data)));

	cudaCheckErrors(hipEventRecord(memcpyStart));
	cudaCheckErrors(hipMemcpy(d_data, h_data, N * L * sizeof(*d_data), hipMemcpyHostToDevice));
	cudaCheckErrors(hipEventRecord(memcpyEnd));

	thrust::pair<thrust::pair<uint_fast64_t, uint_fast64_t>, int>* hashes = nullptr;

	cudaCheckErrors(hipMalloc((void**)&hashes, N * sizeof(*hashes)));

	hipDeviceProp_t prop{};

	cudaCheckErrors(hipGetDeviceProperties(&prop, device));

	int numThreads = prop.maxThreadsPerBlock;
	int numBlocks = (int)ceil((double)N / numThreads);

	cudaCheckErrors(hipEventRecord(computeStart));
	computeHashes << <numBlocks, numThreads >> > (hashes, d_data, N, L);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(computeEnd));

	cudaCheckErrors(hipEventRecord(sortStart));
	// Sort in ascending order (the first element of a pair is compared first)
	thrust::sort(thrust::device, hashes, hashes + N);
	cudaCheckErrors(hipEventRecord(sortEnd));

	cudaCheckErrors(hipEventRecord(findStart));
	findHammingOne << <numBlocks, numThreads >> > (hashes, d_data, N, L);
	cudaCheckErrors(hipGetLastError());
	cudaCheckErrors(hipEventRecord(findEnd));
	cudaCheckErrors(hipEventSynchronize(findEnd));

	float readTime = .0f;
	float memcpyTime = .0f;
	float computeTime = .0f;
	float sortTime = .0f;
	float findTime = .0f;

	cudaCheckErrors(hipEventElapsedTime(&readTime, readStart, readEnd));
	cudaCheckErrors(hipEventElapsedTime(&memcpyTime, memcpyStart, memcpyEnd));
	cudaCheckErrors(hipEventElapsedTime(&computeTime, computeStart, computeEnd));
	cudaCheckErrors(hipEventElapsedTime(&sortTime, sortStart, sortEnd));
	cudaCheckErrors(hipEventElapsedTime(&findTime, findStart, findEnd));

	writeStats(N, L, readTime, memcpyTime, computeTime, sortTime, findTime);

	delete[] h_data;
	cudaCheckErrors(hipEventDestroy(readStart));
	cudaCheckErrors(hipEventDestroy(readEnd));
	cudaCheckErrors(hipEventDestroy(memcpyStart));
	cudaCheckErrors(hipEventDestroy(memcpyEnd));
	cudaCheckErrors(hipEventDestroy(computeStart));
	cudaCheckErrors(hipEventDestroy(computeEnd));
	cudaCheckErrors(hipEventDestroy(sortStart));
	cudaCheckErrors(hipEventDestroy(sortEnd));
	cudaCheckErrors(hipEventDestroy(findStart));
	cudaCheckErrors(hipEventDestroy(findEnd));
	cudaCheckErrors(hipFree(d_data));
	cudaCheckErrors(hipFree(hashes));
	cudaCheckErrors(hipDeviceReset());
	return EXIT_SUCCESS;
}